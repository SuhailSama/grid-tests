#include "hip/hip_runtime.h"
// CUDA Example: "Single-precision A*X Plus Y"
// Derived from:
// https://devblogs.nvidia.com/parallelforall/easy-introduction-cuda-c-and-c/
#include <stdio.h>

// CUDA kernel code, to run on GPU
__global__
void saxpy(int n, float a, float *x, float *y)
{
  // blockDim:  dimensions of each thread block
  // threadIdx: index of this thread within its thread block
  // blockIdx:  index of this thread block within grid
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  // The conditional prevents an out-of-bounds issue if the number of elements
  // in the arrays doesn't fit evenly into the number of thread blocks.  The
  // assignment just carries out the operation element-wise, but it will
  // actually be performed in parallel with many separate calls to this
  // function.
  if (i < n) y[i] = a*x[i] + y[i];
}

// Regular C code, to run on CPU.
// The saxpy() call below will execute on the GPU.
int main(void)
{
  printf("CUDA Example \"Single-precision A*X Plus Y\"\n\n");

  int N = 1<<20;
  int bytesneeded = N*sizeof(float);
  printf("Allocating %d float values (%d MB)...\n", N, bytesneeded>>20);

  float *x, *y, *d_x, *d_y;
  // Arrays allocated in host memory
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  // Arrays allocated in CUDA device memory
  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  printf("Running calculation on GPU...\n");
  // Copy both arrays from the host to the device
  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  //
  // The syntax is:
  // <<< Thread blocks in grid , threads in thread block >>>
  // So, 256 threads per thread block, and however many thread blocks needed to
  // represent the full array.
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  // Copy result from device back to host
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  // Since for every element the answer should be 2*1+2 = 4, this will check
  // our results.
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Done. Max error: %f\n", maxError);
}
